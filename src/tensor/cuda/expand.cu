#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <cassert>
#include <cuda/std/detail/libcxx/include/array>
#include "lamppp/tensor/cuda/expand.cuh"
#include "lamppp/tensor/cuda/kernels.cuh"

namespace lmp::tensor::detail::cuda {

template <typename PtrList, typename OpFn>
__global__ void vectorized_expand_kernel(PtrList ptr_, OpFn fn_, size_t size,
                                         const OffsetUtil<2>* align) {
  size_t i = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (i < size) {
    ::cuda::std::array offsets = align->get(i);
    ptr_.set_Out(i, fn_((ptr_.fns[1])(ptr_.data[1], offsets[1]),
                        (ptr_.fns[2])(ptr_.data[2], offsets[2])));
  }
}

template <typename PtrList, typename OpFn>
void expand_kernel_launcher(PtrList ptr_, OpFn fn_, size_t size,
                            const OffsetUtil<2>* align) {
  size_t threads = 256;
  size_t blocks = (size + threads - 1) / threads;
  ListDevicePtr<OffsetUtil<2>> d_align(align, 1);
  vectorized_expand_kernel<<<blocks, threads>>>(ptr_, fn_, size, d_align.get());

  hipError_t sync_err = hipDeviceSynchronize();
  // assert(sync_err == hipSuccess && "expand_kernel_launcher: kernel failed.");
  if (sync_err != hipSuccess) {
    const char* err_str = hipGetErrorString(sync_err);
    printf("expand_kernel_launcher: kernel failed with error: %s\n", err_str);
    assert(false);
  }
}

template void expand_dispatch_handler<AddFunctor>(
    const internal::TensorMetaHandler&);

}  // namespace lmp::tensor::detail::cuda