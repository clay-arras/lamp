#include "hip/hip_runtime.h"
#include <boost/preprocessor/seq/elem.hpp>
#include <boost/preprocessor/seq/for_each.hpp>
#include <boost/preprocessor/seq/for_each_product.hpp>
#include "lamppp/tensor/cuda/matrix_kern.cuh"

namespace lmp::tensor::detail::cuda {

template <typename U, typename V, typename OutType>
__global__ void cudaMatmulKernel(const U* A, const V* B, OutType* C, size_t m,
                                 size_t n, size_t k) {
  size_t i = threadIdx.x + (blockIdx.x * blockDim.x);
  size_t j = threadIdx.y + (blockIdx.y * blockDim.y);

  if (i < m && j < n) {
    OutType sum = 0;
    for (size_t t = 0; t < k; t++) {
      sum += static_cast<OutType>(A[(i * k) + t]) *
             static_cast<OutType>(B[(n * t) + j]);
    }
    C[(i * n) + j] = sum;
  }
}

template <typename T>
__global__ void cudaTransposeKernel(const T* in, T* out, size_t m, size_t n) {
  size_t i = threadIdx.x + (blockIdx.x * blockDim.x);
  size_t j = threadIdx.y + (blockIdx.y * blockDim.y);

  if (i < m && j < n) {
    out[(j * m) + i] = in[(i * n) + j];
  }
}

template <typename U, typename V, typename OutType>
void cudaMatMul(const U* A, const V* B, OutType* C, size_t m, size_t n,
                size_t k) {
  dim3 threads(16, 16);
  dim3 blocks((m + threads.x - 1) / threads.x, (n + threads.y - 1) / threads.y);
  cudaMatmulKernel<U, V, OutType><<<blocks, threads>>>(A, B, C, m, n, k);
}

template <typename T>
void cudaTranspose(const T* in, T* out, size_t m, size_t n) {
  dim3 threads(16, 16);
  dim3 blocks((m + threads.x - 1) / threads.x, (n + threads.y - 1) / threads.y);
  cudaTransposeKernel<T><<<blocks, threads>>>(in, out, m, n);
}

// clang-format off
#define INSTANTIATE_MATMUL(r, product)                                      \
  template void                                                             \
  cudaMatMul<BOOST_PP_SEQ_ELEM(0, product), BOOST_PP_SEQ_ELEM(1, product),  \
             BOOST_PP_SEQ_ELEM(2, product)>(                                \
      const BOOST_PP_SEQ_ELEM(0, product)*,                                 \
      const BOOST_PP_SEQ_ELEM(1, product)*, BOOST_PP_SEQ_ELEM(2, product)*, \
      size_t, size_t, size_t);

#define INSTANTIATE_TRANSPOSE(r, data, elem) \
  template void cudaTranspose<elem>(const elem*, elem*, size_t, size_t);

#include "lamppp/tensor/supported_types.hpp"
#define TYPES_LIST LMP_TYPES()
BOOST_PP_SEQ_FOR_EACH_PRODUCT(INSTANTIATE_MATMUL,
                              (TYPES_LIST)(TYPES_LIST)(TYPES_LIST))
BOOST_PP_SEQ_FOR_EACH(INSTANTIATE_TRANSPOSE, , TYPES_LIST)

#undef INSTANTIATE_MATMUL
#undef INSTANTIATE_TRANSPOSE
// clang-format on

}  // namespace lmp::tensor::detail::cuda
