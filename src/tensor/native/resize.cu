#include "hip/hip_runtime.h"
#include "include/lamppp/tensor/native/resize.cuh"

#include <hip/hip_runtime.h>
#include <algorithm>
#include <cstring>
#include <new>

namespace lmp::tensor::detail::native {

LMP_DEFINE_DISPATCH(resize_stub);

void resize_cpu(DataPtr dptr, size_t old_byte_size, size_t new_byte_size) {
  void* ptr = ::operator new(new_byte_size);
  std::memcpy(ptr, dptr.data(), std::min(old_byte_size, new_byte_size));

  auto deleter = std::get_deleter<std::function<void(void*)>>(dptr.ptr);
  dptr = DataPtr(ptr, *deleter);
}

void resize_cuda(DataPtr dptr, size_t old_byte_size, size_t new_byte_size) {
  void* ptr = nullptr;
  hipMalloc(&ptr, new_byte_size);
  hipMemcpy(ptr, dptr.data(), std::min(old_byte_size, new_byte_size),
             hipMemcpyDeviceToDevice);

  auto deleter = std::get_deleter<std::function<void(void*)>>(dptr.ptr);
  dptr = DataPtr(ptr, *deleter);
}

}  // namespace lmp::tensor::detail::native
