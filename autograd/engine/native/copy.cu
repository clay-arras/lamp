#include <cstring>
#include "autograd/engine/device_type.hpp"
#include "autograd/engine/dispatch_stub.hpp"
#include "copy.cuh"

namespace autograd {

DEFINE_DISPATCH(copy_stub);

void copy_cpu(const void* src, void* dest, size_t size, DeviceType to_device) {
  switch (to_device) {
    case DeviceType::CPU: {
      memcpy(dest, src, size);
      break;
    }
    case DeviceType::CUDA: {
      hipMemcpy(dest, src, size, hipMemcpyDeviceToHost);
      break;
    }
  }
}

void copy_cuda(const void* src, void* dest, size_t size, DeviceType to_device) {
  switch (to_device) {
    case DeviceType::CPU: {
      hipMemcpy(dest, src, size, hipMemcpyDeviceToHost);
      break;
    }
    case DeviceType::CUDA: {
      hipMemcpy(dest, src, size, hipMemcpyDeviceToDevice);
      break;
    }
  }
}

}  // namespace autograd