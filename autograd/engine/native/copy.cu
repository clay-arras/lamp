#include <cstring>
#include "autograd/engine/device_type.hpp"
#include "autograd/engine/dispatch_stub.hpp"
#include "copy.cuh"

namespace autograd {

DEFINE_DISPATCH(copy_stub);

void copy_cpu(DeviceType to_device, const void* src, void* dest, size_t size,
              DataType src_dtype, DataType dest_dtype) {
  switch (to_device) {
    case DeviceType::CPU: {
      memcpy(dest, src, size);
      break;
    }
    case DeviceType::CUDA: {
      hipMemcpy(dest, src, size, hipMemcpyHostToDevice);
      break;
    }
  }
}

void copy_cuda(DeviceType to_device, const void* src, void* dest, size_t size,
               DataType src_dtype, DataType dest_dtype) {
  switch (to_device) {
    case DeviceType::CPU: {
      hipMemcpy(dest, src, size, hipMemcpyDeviceToHost);
      break;
    }
    case DeviceType::CUDA: {
      hipMemcpy(dest, src, size, hipMemcpyDeviceToDevice);
      break;
    }
  }
}

}  // namespace autograd