#include "hip/hip_runtime.h"
#include "reduct_kern.cuh"

namespace autograd {

inline namespace cuda {

namespace {

__global__ void vecSumKernel(const float* in,
                             float* out,
                             const int* shape,
                             int* stride,
                             int axis, 
                             int outSize) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < outSize) {
        size_t outer = stride[axis];
        size_t inner = stride[axis+1];            
        size_t idx   = (i / outer) * inner
                     + (i % outer);

        float sum = 0.0f;                      
        for (int j = 0; j < shape[axis]; ++j) {
            sum += in[idx + j * outer];           
        }

        out[i] = sum;
    }
}

__global__ void vecMaxKernel(const float* in,
                             float* out,
                             const int* shape,
                             int* stride,
                             int axis, 
                             int outSize) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < outSize) {
        size_t outer = stride[axis];
        size_t inner = stride[axis+1];            
        size_t idx   = (i / outer) * inner
                     + (i % outer);

        float max = 0.0f;                      
        for (int j = 0; j < shape[axis]; ++j) {
            max = fmaxf(max, in[idx + j * outer]); 
        }

        out[i] = max;
    }
}

} // namespace

extern "C" void vecSum(const float* in,
                       float* out,
                       const int* shape,
                       int axis,
                       int ndims) {
    int totalSize = 1;
    for (int i = 0; i < ndims; ++i) {
        totalSize *= shape[i];
    }
    int outSize = totalSize / shape[axis];
    int* h_stride = new int[ndims + 1];

    h_stride[0] = 1;
    for (int i = 1; i <= ndims; i++) {
        h_stride[i] = h_stride[i - 1] * shape[i-1];
    }
    
    float *d_in, *d_out;
    int *d_shape, *d_stride;
    
    hipMalloc(&d_in, totalSize * sizeof(float));
    hipMalloc(&d_out, outSize * sizeof(float));
    hipMalloc(&d_shape, ndims * sizeof(int));
    hipMalloc(&d_stride, (ndims + 1) * sizeof(int));
    
    hipMemcpy(d_in, in, totalSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_shape, shape, ndims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_stride, h_stride, (ndims + 1) * sizeof(int), hipMemcpyHostToDevice);
    
    int threads = 256;
    int blocks = (outSize + threads - 1) / threads;
    vecSumKernel<<<blocks, threads>>>(d_in, d_out, d_shape, d_stride, axis, outSize);
    
    hipMemcpy(out, d_out, outSize * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_shape);
    hipFree(d_stride);
    
    delete[] h_stride;
}

extern "C" void vecMax(const float* in,
                       float* out,
                       const int* shape,
                       int axis, 
                       int ndims) {
    int totalSize = 1;
    for (int i = 0; i < ndims; ++i) {
        totalSize *= shape[i];
    }
    int outSize = totalSize / shape[axis];
    int* h_stride = new int[ndims + 1];

    h_stride[0] = 1;
    for (int i = 1; i <= ndims; i++) {
        h_stride[i] = h_stride[i - 1] * shape[i-1];
    }
    
    float *d_in, *d_out;
    int *d_shape, *d_stride;
    
    hipMalloc(&d_in, totalSize * sizeof(float));
    hipMalloc(&d_out, outSize * sizeof(float));
    hipMalloc(&d_shape, ndims * sizeof(int));
    hipMalloc(&d_stride, (ndims + 1) * sizeof(int));
    
    hipMemcpy(d_in, in, totalSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_shape, shape, ndims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_stride, h_stride, (ndims + 1) * sizeof(int), hipMemcpyHostToDevice);
    
    int threads = 256;
    int blocks = (outSize + threads - 1) / threads;
    vecSumKernel<<<blocks, threads>>>(d_in, d_out, d_shape, d_stride, axis, outSize);
    
    hipMemcpy(out, d_out, outSize * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_shape);
    hipFree(d_stride);
    
    delete[] h_stride;

}

} // namespace cuda

} // namespace autograd