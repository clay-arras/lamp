#include "hip/hip_runtime.h"
#include "matrix_kern.cuh"

namespace autograd {

inline namespace cuda {

// Kernels moved outside anonymous namespace
template <typename T>
__global__ void cudaMatmulKernel(const T* A, const T* B, T* C, int m, int n, int k) {
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    int j = threadIdx.y + (blockIdx.y * blockDim.y);

    if (i < m && j < n) {
        T sum = 0;
        for (int t=0; t<k; t++) { // NOTE: A is MxK, B is KxN, C is MxN
            // sum += A[(i*k) + t] * B[(n*t) + j]; // row major implementation
            sum += A[i + m*t] * B[t + k*j]; // TODO(nlin): this can be made faster but whatever
        }
        C[(j*m) + i] = sum;
    }
}

template <typename T>
__global__ void cudaTransposeKernel(const T* in,
                                    T* out,
                                    int m,
                                    int n) {
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    int j = threadIdx.y + (blockIdx.y * blockDim.y);

    if (i < m && j < n) {
        out[(i*n) + j] = in[(j*m) + i];
    }
}

template <typename T>
void cudaMatMul(const T* A, const T* B, T* C, int m, int n, int k) {
  T *d_a;
  T *d_b;
  T *d_c;
  size_t bytes_a = m * k * sizeof(T);
  size_t bytes_b = k * n * sizeof(T);
  size_t bytes_c = m * n * sizeof(T);

  hipMalloc(&d_a, bytes_a);
  hipMalloc(&d_b, bytes_b);
  hipMalloc(&d_c, bytes_c);
  hipMemcpy(d_a, A, bytes_a, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, bytes_b, hipMemcpyHostToDevice);

  dim3 threads(16, 16);
  dim3 blocks((m + threads.x - 1) / threads.x, (n + threads.y - 1) / threads.y);
  cudaMatmulKernel<<<blocks, threads>>>(d_a, d_b, d_c, m, n, k);

  hipMemcpy(C, d_c, bytes_c, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

template <typename T>
void cudaTranspose(const T* in,
                              T* out,
                              int m,
                              int n) {
  T *d_in;
  T *d_out;
  size_t bytes_in = m * n * sizeof(T);
  size_t bytes_out = m * n * sizeof(T);

  hipMalloc(&d_in, bytes_in);
  hipMalloc(&d_out, bytes_out);
  hipMemcpy(d_in, in, bytes_in, hipMemcpyHostToDevice);

  dim3 threads(16, 16);
  dim3 blocks((m + threads.x - 1) / threads.x, (n + threads.y - 1) / threads.y);
  cudaTransposeKernel<<<blocks, threads>>>(d_in, d_out, m, n);

  hipMemcpy(out, d_out, bytes_out, hipMemcpyDeviceToHost);

  hipFree(d_in);
  hipFree(d_out);
}

#define X(TYPE) template void cudaMatMul<TYPE>(const TYPE*, const TYPE*, TYPE*, int, int, int); \
                 template void cudaTranspose<TYPE>(const TYPE*, TYPE*, int, int);
#include "autograd/engine/supported_types.def"
#undef  X

} // namespace cuda

} // namespace autograd