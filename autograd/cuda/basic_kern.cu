#include "hip/hip_runtime.h"
namespace autograd {

namespace cuda {

namespace {

__global__ void add(int size, const float* A, const float* B, float* C) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = A[i] + B[i];
    }
}

void vecAdd(int size, const float* A, const float* B, float* C) {
  float *d_a, *d_b, *d_out;
  size_t bytes = n * sizeof(float);

  hipMalloc(&d_a,    bytes);
  hipMalloc(&d_b,    bytes);
  hipMalloc(&d_out,  bytes);

  hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);

  int threads = 256;
  int blocks  = (n + threads - 1) / threads;
  vectorAddKernel<<<blocks, threads>>>(d_a, d_b, d_out, n);

  hipMemcpy(out, d_out, bytes, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_out);
}

__global__ void sub(int size, const float* A, const float* B, float* C) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = A[i] - B[i];
    }
}

__global__ void mul(int size, const float* A, const float* B, float* C) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = A[i] * B[i];
    }
}

__global__ void div(int size, const float* A, const float* B, float* C) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = A[i] / B[i];
    }
}

}

}

}