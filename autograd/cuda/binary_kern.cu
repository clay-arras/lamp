#include "hip/hip_runtime.h"
#include "binary_kern.cuh"

namespace autograd {

inline namespace cuda {

template <typename T>
__global__ void vecEqualKernel(size_t size, const T* A, const T* B, T* C) {
    size_t i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = (A[i] == B[i]) ? 1.0F : 0.0F;
    }
}

template <typename T>
__global__ void vecNotEqualKernel(size_t size, const T* A, const T* B, T* C) {
    size_t i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = (A[i] != B[i]) ? 1.0F : 0.0F;
    }
}

template <typename T>
__global__ void vecGreaterEqualKernel(size_t size, const T* A, const T* B, T* C) {
    size_t i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = (A[i] >= B[i]) ? 1.0F : 0.0F;
    }
}

template <typename T>
__global__ void vecLessEqualKernel(size_t size, const T* A, const T* B, T* C) {
    size_t i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = (A[i] <= B[i]) ? 1.0F : 0.0F;
    }
}

template <typename T>
__global__ void vecGreaterThanKernel(size_t size, const T* A, const T* B, T* C) {
    size_t i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = (A[i] > B[i]) ? 1.0F : 0.0F;
    }
}

template <typename T>
__global__ void vecLessThanKernel(size_t size, const T* A, const T* B, T* C) {
    size_t i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = (A[i] < B[i]) ? 1.0F : 0.0F;
    }
}

template <typename T>
void vecEqual(size_t size, const T* A, const T* B, T* C) {
  T *d_a;
  T *d_b;
  T *d_c;
  size_t bytes = size * sizeof(T);

  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, bytes, hipMemcpyHostToDevice);

  size_t threads = 256;
  size_t blocks = (size + threads - 1) / threads;
  vecEqualKernel<<<blocks, threads>>>(size, d_a, d_b, d_c);

  hipMemcpy(C, d_c, bytes, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

template <typename T>
void vecNotEqual(size_t size, const T* A, const T* B, T* C) {
  T *d_a;
  T *d_b;
  T *d_c;
  size_t bytes = size * sizeof(T);

  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, bytes, hipMemcpyHostToDevice);

  size_t threads = 256;
  size_t blocks = (size + threads - 1) / threads;
  vecNotEqualKernel<<<blocks, threads>>>(size, d_a, d_b, d_c);

  hipMemcpy(C, d_c, bytes, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

template <typename T>
void vecGreaterEqual(size_t size, const T* A, const T* B, T* C) {
  T *d_a;
  T *d_b;
  T *d_c;
  size_t bytes = size * sizeof(T);

  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, bytes, hipMemcpyHostToDevice);

  size_t threads = 256;
  size_t blocks = (size + threads - 1) / threads;
  vecGreaterEqualKernel<<<blocks, threads>>>(size, d_a, d_b, d_c);

  hipMemcpy(C, d_c, bytes, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

template <typename T>
void vecLessEqual(size_t size, const T* A, const T* B, T* C) {
  T *d_a;
  T *d_b;
  T *d_c;
  size_t bytes = size * sizeof(T);

  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, bytes, hipMemcpyHostToDevice);

  size_t threads = 256;
  size_t blocks = (size + threads - 1) / threads;
  vecLessEqualKernel<<<blocks, threads>>>(size, d_a, d_b, d_c);

  hipMemcpy(C, d_c, bytes, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

template <typename T>
void vecGreaterThan(size_t size, const T* A, const T* B, T* C) {
  T *d_a;
  T *d_b;
  T *d_c;
  size_t bytes = size * sizeof(T);

  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, bytes, hipMemcpyHostToDevice);

  size_t threads = 256;
  size_t blocks = (size + threads - 1) / threads;
  vecGreaterThanKernel<<<blocks, threads>>>(size, d_a, d_b, d_c);

  hipMemcpy(C, d_c, bytes, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

template <typename T>
void vecLessThan(size_t size, const T* A, const T* B, T* C) {
  T *d_a;
  T *d_b;
  T *d_c;
  size_t bytes = size * sizeof(T);

  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, bytes, hipMemcpyHostToDevice);

  size_t threads = 256;
  size_t blocks = (size + threads - 1) / threads;
  vecLessThanKernel<<<blocks, threads>>>(size, d_a, d_b, d_c);

  hipMemcpy(C, d_c, bytes, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

#define X(TYPE) template void vecEqual<TYPE>(size_t, const TYPE*, const TYPE*, TYPE*); \
                 template void vecNotEqual<TYPE>(size_t, const TYPE*, const TYPE*, TYPE*); \
                 template void vecGreaterEqual<TYPE>(size_t, const TYPE*, const TYPE*, TYPE*); \
                 template void vecLessEqual<TYPE>(size_t, const TYPE*, const TYPE*, TYPE*); \
                 template void vecGreaterThan<TYPE>(size_t, const TYPE*, const TYPE*, TYPE*); \
                 template void vecLessThan<TYPE>(size_t, const TYPE*, const TYPE*, TYPE*);
#include "autograd/engine/supported_types.def"
#undef  X

} // namespace cuda

} // namespace autograd