#include "hip/hip_runtime.h"
#include "binary_kern.cuh"

namespace autograd {

inline namespace cuda {

template <typename T>
__global__ void vecEqualKernel(int size, const T* A, const T* B, T* C) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = (A[i] == B[i]) ? 1.0F : 0.0F;
    }
}

template <typename T>
__global__ void vecNotEqualKernel(int size, const T* A, const T* B, T* C) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = (A[i] != B[i]) ? 1.0F : 0.0F;
    }
}

template <typename T>
__global__ void vecGreaterEqualKernel(int size, const T* A, const T* B, T* C) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = (A[i] >= B[i]) ? 1.0F : 0.0F;
    }
}

template <typename T>
__global__ void vecLessEqualKernel(int size, const T* A, const T* B, T* C) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = (A[i] <= B[i]) ? 1.0F : 0.0F;
    }
}

template <typename T>
__global__ void vecGreaterThanKernel(int size, const T* A, const T* B, T* C) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = (A[i] > B[i]) ? 1.0F : 0.0F;
    }
}

template <typename T>
__global__ void vecLessThanKernel(int size, const T* A, const T* B, T* C) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = (A[i] < B[i]) ? 1.0F : 0.0F;
    }
}

template <typename T>
void vecEqual(int size, const T* A, const T* B, T* C) {
  T *d_a;
  T *d_b;
  T *d_c;
  size_t bytes = size * sizeof(T);

  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, bytes, hipMemcpyHostToDevice);

  int threads = 256;
  int blocks = (size + threads - 1) / threads;
  vecEqualKernel<<<blocks, threads>>>(size, d_a, d_b, d_c);

  hipMemcpy(C, d_c, bytes, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

template <typename T>
void vecNotEqual(int size, const T* A, const T* B, T* C) {
  T *d_a;
  T *d_b;
  T *d_c;
  size_t bytes = size * sizeof(T);

  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, bytes, hipMemcpyHostToDevice);

  int threads = 256;
  int blocks = (size + threads - 1) / threads;
  vecNotEqualKernel<<<blocks, threads>>>(size, d_a, d_b, d_c);

  hipMemcpy(C, d_c, bytes, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

template <typename T>
void vecGreaterEqual(int size, const T* A, const T* B, T* C) {
  T *d_a;
  T *d_b;
  T *d_c;
  size_t bytes = size * sizeof(T);

  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, bytes, hipMemcpyHostToDevice);

  int threads = 256;
  int blocks = (size + threads - 1) / threads;
  vecGreaterEqualKernel<<<blocks, threads>>>(size, d_a, d_b, d_c);

  hipMemcpy(C, d_c, bytes, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

template <typename T>
void vecLessEqual(int size, const T* A, const T* B, T* C) {
  T *d_a;
  T *d_b;
  T *d_c;
  size_t bytes = size * sizeof(T);

  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, bytes, hipMemcpyHostToDevice);

  int threads = 256;
  int blocks = (size + threads - 1) / threads;
  vecLessEqualKernel<<<blocks, threads>>>(size, d_a, d_b, d_c);

  hipMemcpy(C, d_c, bytes, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

template <typename T>
void vecGreaterThan(int size, const T* A, const T* B, T* C) {
  T *d_a;
  T *d_b;
  T *d_c;
  size_t bytes = size * sizeof(T);

  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, bytes, hipMemcpyHostToDevice);

  int threads = 256;
  int blocks = (size + threads - 1) / threads;
  vecGreaterThanKernel<<<blocks, threads>>>(size, d_a, d_b, d_c);

  hipMemcpy(C, d_c, bytes, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

template <typename T>
void vecLessThan(int size, const T* A, const T* B, T* C) {
  T *d_a;
  T *d_b;
  T *d_c;
  size_t bytes = size * sizeof(T);

  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, bytes, hipMemcpyHostToDevice);

  int threads = 256;
  int blocks = (size + threads - 1) / threads;
  vecLessThanKernel<<<blocks, threads>>>(size, d_a, d_b, d_c);

  hipMemcpy(C, d_c, bytes, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

// Explicit template instantiations
template void vecEqual<float>(int size, const float* A, const float* B, float* C);
template void vecNotEqual<float>(int size, const float* A, const float* B, float* C);
template void vecGreaterEqual<float>(int size, const float* A, const float* B, float* C);
template void vecLessEqual<float>(int size, const float* A, const float* B, float* C);
template void vecGreaterThan<float>(int size, const float* A, const float* B, float* C);
template void vecLessThan<float>(int size, const float* A, const float* B, float* C);

} // namespace cuda

} // namespace autograd