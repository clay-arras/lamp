
#include <hip/hip_runtime.h>
namespace autograd {

namespace {

__global__ void add(int size, const float* A, const float* B, float* C) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = A[i] + B[i];
    }
}

__global__ void sub(int size, const float* A, const float* B, float* C) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = A[i] - B[i];
    }
}

__global__ void mul(int size, const float* A, const float* B, float* C) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = A[i] * B[i];
    }
}

__global__ void div(int size, const float* A, const float* B, float* C) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        C[i] = A[i] / B[i];
    }
}

}

}